#include "hip/hip_runtime.h"
/*
    A C++ implementation of Lenstra's algorithm, adapted from that from Hua Li:
        https://researchportal.bath.ac.uk/en/publications/the-analysis-and-implementation-of-the-aks-algorithm-and-its-impr
    Compile with:
        $ g++ -g -O2 -std=c++11 -pthread -march=native dir/foo.cpp -o dir/foo.out -lntl -lgmp -lm
        $ nvcc -o HPCLenstraZnx.out HPCLenstraZnx.cu
        $ nvcc HPCLenstraZnx.cu -o HPCLenstraZnx.out
*/

#include <math.h> // standard libraries
#include <fstream>
#include <iostream>
#include <sstream>
#include <iomanip>
// #include <windows.h>
#include <unistd.h>
#include <stdio.h>
#include <cstdio>
#include <stdlib.h>
#include <cstdlib>
// #include <mmsystem.h>
#include <time.h>
#include <ctime>
#include <chrono>
#include <string>
#include <thread>
#include <sys/time.h>
#include <sys/resource.h>

#include "NTL/ZZ.h" // NTL Libraries
#include "NTL/ZZ_p.h"
#include "NTL/ZZ_pX.h"
#include "NTL/ZZX.h"
#include "NTL/vec_ZZ.h"
#include <NTL/ZZ_pXFactoring.h>
#include <NTL/BasicThreadPool.h>
NTL_CLIENT

// includes CUDA
#include <hip/hip_runtime.h>

// includes, project
#include <hip/hip_runtime_api.h>
#include <helper_functions.h> // helper functions for SDK examples

#include "PerfectPower.h" //Each Indepedent Test
#include "Euler.h"

std::string getTime() {
    auto t = std::time(nullptr);
    auto tm = *std::localtime(&t);

    std::ostringstream oss;
    oss << std::put_time(&tm, "%Y-%m-%d-%H-%M-%S");
    auto time = oss.str();

    return time;
}

std::string getFilename() {
    std::string prfx = "log-LenstraZ-";
    std::string sffx = getTime();
    std::string extn = ".csv";

    std::string filename = prfx + sffx + extn;

    return filename;
}

unsigned int ncores = std::thread::hardware_concurrency(); // machine cores - may return 0 when not able to detect
const auto SetNumThreads(ncores); // number of threads - should correspond to the number of available cores on your machine

std::string filename = getFilename();
std::ofstream perflog(filename, std::ios::app); // output result into file

inline void fileWrite(const ZZ& n, const unsigned int& cores, const bool& PRIME, const long& time, const std::string& other) {
    perflog << n << "," << cores << "," << PRIME  << "," << time << "," << other << "\n";
}

__global__ void CongruenceZnx (ZZ *n, ZZ *r, ZZ *r2) { // __global__ void kernel(ZZ *d_n, ZZ *d_r, ZZ *d_r2){
    // congruence test of polynomials in regular form

    // Thread indexing
    int i = threadIdx.x;

    // Perform this operation for every thread

    for(long a = 1; a <= to_long(*r2 - 1); ++a){
        ZZ_p::init(*n); //mod n
        ZZ_pX b = ZZ_pX(to_long(*r), 1) - 1; // b = x^r - 1;
        ZZ_pX c = ZZ_pX(1, 1) - a ; // c = x - a;
        ZZ_pX f = PowerMod(c, *n, b); // f =(x - a)^n mod c, n which is the RHS
        ZZ_pX e = ZZ_pX(1, 1);
        ZZ_pX g = PowerMod(e, *n, b); // x^n mod b, n
        g = g - a ; // g1 = x^n - a mod c, n.

        if(f == g){
            return(1); // n is prime
        }
        else{
            return(a); // n is not prime.
        }
    }
}

inline bool Lenstra (const ZZ& n) {
    if(n < 1){
        std::printf("Integer n needs to be positive.\n");
        return false;
    }
    else if(n == 1){
        std::printf("1 is neither prime or composite.\n");
        return false;
    }
    else if(n == 2){
        std::printf("2 is prime.\n");
        return true;
    }
    else if(n == 3){
        std::printf("3 is prime.\n");
        return true;
    }

    std::printf("n = %ld\n",to_long(n));

    // start timing
    auto start = std::chrono::steady_clock::now();

    // Test if n is a perfect power
    int PP = PerfectPower(n);

    // returns 1 if n is a perfect power, 0 otherwise;
    if(PP == 1){
        auto finish = std::chrono::steady_clock::now();
        auto duration = finish - start;
        auto time = std::chrono::duration_cast<std::chrono::milliseconds>(duration).count();

        std::printf("%ld is not prime.\n",to_long(n));
        std::printf("%ld is a perfect power.\n",to_long(n));
        std::printf("Time taken: %ld milliseconds\n",time);

        std::string note = std::to_string(to_long(n)) + " is a perfect power";
        fileWrite(n,ncores,false,time,note);

        return false;
    }

    // Find a suitable r
    ZZ r = to_ZZ(2);
    ZZ R;
    ZZ r1;

    while(r < n){
        ZZ R = GCD(r, n);
        if(R != 1 ){
            auto finish = std::chrono::steady_clock::now();
            auto duration = finish - start;
            auto time = std::chrono::duration_cast<std::chrono::milliseconds>(duration).count();

            std::printf("%ld is not prime.\n",to_long(n));
            std::printf("%ld is a divisor.\n",to_long(R));
            std::printf("Time taken: %ld milliseconds\n",time);

            std::string note = std::to_string(to_long(R)) + " is a divisor";
            fileWrite(n,ncores,false,time,note);

            return false;
            break;
        }
        else {
            ZZ v = to_ZZ(floor(power_long(to_long(log(n)), 2)));

            // order of n mod r is bigger than v;
            int p = 0;
            ZZ_p::init(r); // calculate mod r

            while(v <= r){
                ZZ x = to_ZZ(power_long(to_long(n), to_long(v))); // calculates x = n^v
                ZZ_p z = to_ZZ_p(x);
                if(z == to_ZZ_p(1)){
                    r1 = r; // store value of r
                    r = n + 1;
                    break;
                }
                else{
                    v = v + 1;
                }
            }
        }
        r = r + 1;
    }

    r = r1;
    std::printf("r = %ld\n",to_long(r));

    ZZ r2 = Euler(to_long(r));
    std::printf("Euler(%ld) = %ld\n",to_long(r),to_long(r2));

    // Declare variables
    int *h_f, *d_f;

    // Allocate memory on the device -- hipMalloc(Location of Memory on Device,sizeof(int));
    hipMalloc((void**)&d_f,sizeof(int));

    // Copy data from Host to Device
    hipMemcpy(d_f,h_f,sizeof(int),hipMemcpyHostToDevice);

    // Configuration Parameters
    dim3 grid_size(1);
    dim3 block_size(N); \\ N threads in block

    // Launch Kernel -- CongruenceZnx<<<grid_size,block_size>>>(d_n,d_r,d_r2)
    CongruenceZnx<<<grid_size,block_size>>>(n,r,r2);

    // Copy data back to host
    hipMemcpy(h_f,d_f,sizeof(int),hipMemcpyDeviceToHost);

    // De-allocate memory
    hipFree(d_f);
    free(h_f);

    if(f == 1){
        auto finish = std::chrono::steady_clock::now();
        auto duration = finish - start;
        auto time = std::chrono::duration_cast<std::chrono::milliseconds>(duration).count();

        std::printf("%ld is prime.\n",to_long(n));
        std::printf("Time taken: %ld milliseconds\n",time);

        std::string note = "n/a";
        fileWrite(n,ncores,true,time,note);

        return true;
    }
    else {
        auto finish = std::chrono::steady_clock::now();
        auto duration = finish - start;
        auto time = std::chrono::duration_cast<std::chrono::milliseconds>(duration).count();

        std::printf("%ld is not prime.\n",to_long(n));
        std::printf("The a which fails is %ld\n",f);
        std::printf("Time taken: %ld milliseconds\n",time);

        std::string note = "a = " + std::to_string(f) + "; r = " + std::to_string(to_long(r)) + "; phi(r) = " + std::to_string(to_long(r2));
        fileWrite(n,ncores,false,time,note);

        return false;
        // break;
    }

}

int main (int argc, char * argv[]) {

    perflog << "Int, Cores, Prime (T/F), Time (milliseconds), Comments\n";

    bool prime;
    ZZ n;
    n = 0;

    std::printf("Enter a positive integer number n you want to be tested:\n");
    std::cin >> n;

    prime = Lenstra(n);

    return 0;
}