#include "hip/hip_runtime.h"
/*
    A C++ implementation of Lenstra's algorithm, adapted from that from Hua Li:
        https://researchportal.bath.ac.uk/en/publications/the-analysis-and-implementation-of-the-aks-algorithm-and-its-impr
    Compile with:
        $ g++ -g -O2 -std=c++11 -pthread -march=native dir/foo.cpp -o dir/foo.out -lntl -lgmp -lm
        $ nvcc -o devVR/LenstraZnx.out devVR/LenstraZnx.cu
        $ nvcc devVR/LenstraZnx.cu -o devVR/LenstraZnx.out
*/

#include <math.h> // standard libraries
#include <fstream>
#include <iostream>
#include <sstream>
#include <iomanip>
// #include <windows.h>
#include <unistd.h>
#include <stdio.h>
#include <cstdio>
#include <stdlib.h>
#include <cstdlib>
#include <filesystem>
// #include <mmsystem.h>
#include <time.h>
#include <ctime>
#include <chrono>
#include <string>
#include <thread>
#include <array>
#include <sys/time.h>
#include <sys/resource.h>
#include <sys/stat.h>
#include <sys/types.h>

#include "NTL/ZZ.h" // NTL Libraries
#include "NTL/ZZ_p.h"
#include "NTL/ZZ_pX.h"
#include "NTL/ZZX.h"
#include "NTL/vec_ZZ.h"
#include <NTL/ZZ_pXFactoring.h>
#include <NTL/BasicThreadPool.h>
NTL_CLIENT

// includes CUDA
#include <hip/hip_runtime.h>

// includes, project
#include <hip/hip_runtime_api.h>
#include <helper_functions.h> // helper functions for SDK examples

#include "PerfectPower.h" //Each Independent Test
#include "Euler.h"

std::string getDate() {
    auto t = std::time(nullptr);
    auto tm = *std::localtime(&t);

    std::ostringstream oss;
    oss << std::put_time(&tm, "%Y-%m-%d");
    auto date = oss.str();

    return date;
}

std::string getTime() {
    auto t = std::time(nullptr);
    auto tm = *std::localtime(&t);

    std::ostringstream oss;
    oss << std::put_time(&tm, "%H-%M-%S");
    auto time = oss.str();

    return time;
}

std::string getDateTime() {
    auto t = std::time(nullptr);
    auto tm = *std::localtime(&t);

    std::ostringstream oss;
    oss << std::put_time(&tm, "%Y-%m-%d-%H-%M-%S");
    auto datetime = oss.str();

    return datetime;
}

std::string getFilename() {
    // std::string make = "/logs/" + getDate();
    // int result = mkdir(make.c_str(), 0777);
    std::string fldr = "logs/HPCznx/";
    // std::string fldr = "logs/" + getDate() + "/";
    std::string prfx = "LnstrZnx-";
    std::string sffx = getDateTime();
    std::string extn = ".csv";

    std::string filename = fldr + prfx + sffx + extn;

    return filename;
}

// std::filesystem::create_directory("logs/" + getDate());
unsigned int ncores = std::thread::hardware_concurrency(); // machine cores - may return 0 when not able to detect
const auto SetNumThreads(ncores); // number of threads - should correspond to the number of available cores on your machine

std::string filename = getFilename();
std::ofstream perflog(filename, std::ios::app); // output result into file

inline void fileWrite(const ZZ& n, const unsigned int& cores, const bool& PRIME, const long& time, const std::string& other) {
    perflog << n << "," << cores << "," << PRIME  << "," << time << "," << other << "\n";
}

__global__ void CongruenceZnx (ZZ *n, ZZ *r, ZZ *r2, long *u, long a) { // __global__ void kernel(ZZ *d_n, ZZ *d_r, ZZ *d_r2){
    // congruence test of polynomials in regular form

    // Thread indexing
    long i = to_long(threadIdx.x);

    // Perform this operation for every thread
    if (i < a) {                            // ensures kernel does not execute more threads than size of a
        ZZ_p::init(n);                      // initialise mod n

        ZZ_pX b = ZZ_pX(to_long(r), 1) - 1; // b = x^r - 1 (mod n);
        ZZ_pX e = ZZ_pX(1, 1);              // e = x (mod n)
        ZZ_pX d = PowerMod(e, n, b);        // d = x^n (mod b, n)

        ZZ_pX c = ZZ_pX(1, 1) - i;          // c = x - a (mod n);
        ZZ_pX f = PowerMod(c, n, b);        // f = (x - a)^n (mod b, n) - LHS
        ZZ_pX g = d - i;                    // g = x^n - a (mod b, n) - RHS

        if(f != g){
            u[i] = 0; // n is not prime
        }
    }
}

inline bool Lenstra (const ZZ& n) {
    if(n < 1){
        std::printf("Integer n needs to be positive.\n");
        return false;
    }
    else if(n == 1){
        std::printf("1 is neither prime or composite.\n");
        return false;
    }
    else if(n == 2){
        std::printf("2 is prime.\n");
        return true;
    }
    else if(n == 3){
        std::printf("3 is prime.\n");
        return true;
    }

    std::cout << "n = " << n << "\n\n";

    // start timing
    auto start = std::chrono::steady_clock::now();

    // Test if n is a perfect power
    int PP = PerfectPower(n);

    // returns 1 if n is a perfect power, 0 otherwise;
    if(PP == 1){
        auto finish = std::chrono::steady_clock::now();
        auto duration = finish - start;
        auto time = std::chrono::duration_cast<std::chrono::milliseconds>(duration).count();

        std::printf("%ld is not prime.\n",to_long(n));
        std::printf("%ld is a perfect power.\n",to_long(n));
        std::printf("Time taken: %ld milliseconds\n",time);

        std::string note = std::to_string(to_long(n)) + " is a perfect power";
        fileWrite(n,ncores,false,time,note);

        return false;
    }

    // Find a suitable r
    ZZ r = to_ZZ(2);
    ZZ R;
    ZZ r1;

    while(r < n){
        ZZ R = GCD(r, n);
        if(R != 1 ){
            auto finish = std::chrono::steady_clock::now();
            auto duration = finish - start;
            auto time = std::chrono::duration_cast<std::chrono::milliseconds>(duration).count();

            std::printf("%ld is not prime.\n",to_long(n));
            std::printf("%ld is a divisor.\n",to_long(R));
            std::printf("Time taken: %ld milliseconds\n",time);

            std::string note = std::to_string(to_long(R)) + " is a divisor";
            fileWrite(n,ncores,false,time,note);

            return false;
            break;
        }
        else {
            ZZ v = to_ZZ(floor(power_long(to_long(log(n)), 2)));

            // order of n mod r is bigger than v;
            int p = 0;
            ZZ_p::init(r); // calculate mod r

            while(v <= r){
                ZZ x = to_ZZ(power_long(to_long(n), to_long(v))); // calculates x = n^v
                ZZ_p z = to_ZZ_p(x);
                if(z == to_ZZ_p(1)){
                    r1 = r; // store value of r
                    r = n + 1;
                    break;
                }
                else{
                    v = v + 1;
                }
            }
        }
        r = r + 1;
    }

    h_r = r1;
    std::printf("r = %ld\n",to_long(h_r));

    ZZ h_r2 = Euler(to_long(h_r));
    std::printf("Euler(%ld) = %ld\n",to_long(h_r),to_long(h_r2));

    // Declare variables
    long h_an;
    long h_av[a];
    ZZ h_n = n;

    long* d_an;
    long* d_av;
    ZZ* d_n;
    ZZ* d_r;
    ZZ* d_r2;

    // Initialise variables
    h_an = to_long(h_r2 - 1);
    for (long x = 0; x < a; ++x) {
        h_av[x] = x + 1;
    }

    // Allocate memory on the device -- hipMalloc(Location of Memory on Device,sizeof(int));
    hipMalloc((void**)&d_an,sizeof(long));
    hipMalloc((void**)&d_av,h_an*sizeof(long));
    hipMalloc((void**)&d_n,sizeof(ZZ));
    hipMalloc((void**)&d_r,sizeof(ZZ));
    hipMalloc((void**)&d_r2,sizeof(ZZ));

    // Copy data from Host to Device
    hipMemcpy(d_an,h_an,sizeof(long),hipMemcpyHostToDevice);
    hipMemcpy(d_av,h_av,h_an*sizeof(long),hipMemcpyHostToDevice);
    hipMemcpy(d_n,h_n,sizeof(ZZ),hipMemcpyHostToDevice);
    hipMemcpy(d_r,h_r,sizeof(ZZ),hipMemcpyHostToDevice);
    hipMemcpy(d_r2,h_r2,sizeof(ZZ),hipMemcpyHostToDevice);

    // Configuration Parameters
    dim3 grid_size(1);
    dim3 block_size(h_an); // a threads in block

    // Launch Kernel -- CongruenceZnx<<<grid_size,block_size>>>(d_n,d_r,d_r2,d_av,d_an)
    CongruenceZnx<<<grid_size,block_size>>>(d_n,d_r,d_r2,d_av,d_an);

    // Copy data back to host
    hipMemcpy(h_av,d_av,h_an*sizeof(long),hipMemcpyDeviceToHost);

    // De-allocate memory
    hipFree(d_an);
    hipFree(d_av);
    hipFree(d_n);
    hipFree(d_r);
    hipFree(d_r2);
    // free(h_av);

    for (long x = 0; x < h_an; ++x) {
        if(h_av[x] == 0){
            auto finish = std::chrono::steady_clock::now();
            auto duration = finish - start;
            auto time = std::chrono::duration_cast<std::chrono::milliseconds>(duration).count();

            long a = x + 1;
            std::printf("%ld is not prime.\n",to_long(n));
            std::printf("The a which fails is %ld\n",a);
            std::printf("Time taken: %ld milliseconds\n",time);

            std::string note = "a = " + std::to_string(a) + "; r = " + std::to_string(to_long(r)) + "; phi(r) = " + std::to_string(to_long(r2));
            fileWrite(n,ncores,false,time,note);

            return false;
            // break;
        }
    }

    auto finish = std::chrono::steady_clock::now();
    auto duration = finish - start;
    auto time = std::chrono::duration_cast<std::chrono::milliseconds>(duration).count();

    std::printf("%ld is prime.\n",to_long(n));
    std::printf("Time taken: %ld milliseconds\n",time);

    std::string note = "n/a";
    fileWrite(n,ncores,true,time,note);

    return true;
}

int main (int argc, char * argv[]) {

    perflog << "Int, Cores, Prime (T/F), Time (milliseconds), Comments\n";

    bool prime;
    ZZ n;
    n = 0;

    std::printf("Enter a positive integer number n you want to be tested:\n");
    std::cin >> n;

    prime = Lenstra(n);

    return 0;
}